#include "hip/hip_runtime.h"
// This file is part of CuPy-RayTracing.
// Copyright (c) 2025, Eijiro Shibusawa <phd_kimberlite@yahoo.co.jp>
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this
//    list of conditions and the following disclaimer.
// 2. Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
// ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
// WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR
// ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
// (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
// LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
// ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

extern "C" __global__ void getPointerSize(int *output)
{
  if (threadIdx.x == 0 && blockIdx.x == 0)
  {
    *output = sizeof(hittable **);
  }
}

extern "C" __global__ void createWorld(unsigned long *objects_ptr, unsigned long *world_ptr)
{
  if (threadIdx.x == 0 && blockIdx.x == 0)
  {
    hittable *objects[2];
    objects[0] = new sphere(point3(0, 0, -1), 0.5f);
    objects[1] = new sphere(point3(0, -100.5f, -1), 100);

    objects_ptr[0] = reinterpret_cast<unsigned long>(objects[0]);
    objects_ptr[1] = reinterpret_cast<unsigned long>(objects[1]);
    world_ptr[0] = reinterpret_cast<unsigned long>(new hittable_list(objects, 2));
  }
}

extern "C" __global__ void destroyWorld(unsigned long *objects_ptr, unsigned long *world_ptr)
{
  if (threadIdx.x == 0 && blockIdx.x == 0)
  {
    hittable *objects = reinterpret_cast<hittable *>(objects_ptr[0]);
    delete objects;
    objects = reinterpret_cast<hittable *>(objects_ptr[1]);
    delete objects;
    objects_ptr[0] = 0;
    objects_ptr[1] = 0;

    hittable *world = reinterpret_cast<hittable *>(world_ptr[0]);
    delete world;
    world_ptr[0] = 0;
  }
}
