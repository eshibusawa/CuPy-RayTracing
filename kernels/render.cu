#include "hip/hip_runtime.h"
// This file is part of CuPy-RayTracing.
// Copyright (c) 2025, Eijiro Shibusawa <phd_kimberlite@yahoo.co.jp>
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this
//    list of conditions and the following disclaimer.
// 2. Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
// ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
// WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR
// ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
// (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
// LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
// ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

__constant__ vec3 g_cameraCenter;
__constant__ vec3 g_pixelDeltaU;
__constant__ vec3 g_pixelDeltaV;
__constant__ vec3 g_pixel00Loc;

using color = vec3;
using point3 = vec3;

__device__ float hit_sphere(const point3& center, double radius, const ray& r)
{
  vec3 oc = center - r.origin();
  auto a = r.direction().length_squared();
  auto h = dot(r.direction(), oc);
  auto c = oc.length_squared() - radius*radius;
  auto discriminant = h*h - a*c;
  if (discriminant < 0)
  {
    return -1.0f;
  }
  else
  {
    return (h - sqrtf(discriminant)) / a;
  }
}

__device__ vec3 ray_color(const ray& r)
{
  auto t = hit_sphere(point3(0, 0, -1), 0.5f, r);
  if (t > 0.0)
  {
    vec3 N = unit_vector(r.at(t) - vec3(0, 0, -1));
    return 0.5f * color(N.x()+1, N.y()+1, N.z()+1);
  }

  vec3 unit_direction = unit_vector(r.direction());
  float a = 0.5f * (unit_direction.y() + 1.0f);
  return (1.0f - a) * vec3(1.0f, 1.0f, 1.0f) + a *vec3(0.5f, 0.7f, 1.0f);
}

extern "C" __global__ void render(vec3 *output)
{
  const int indexX = threadIdx.x + blockIdx.x * blockDim.x;
  const int indexY = threadIdx.y + blockIdx.y * blockDim.y;
  if((indexX >= (RTOW_WIDTH)) || (indexY >= (RTOW_HEIGHT)))
  {
    return;
  }
  auto pixelCenter = g_pixel00Loc + (float(indexX) * g_pixelDeltaU) + (float(indexY) * g_pixelDeltaV);
  auto rayDirection = pixelCenter - g_cameraCenter;
  ray r(g_cameraCenter, rayDirection);

  const int index = indexY * (RTOW_WIDTH) + indexX;
  output[index] = ray_color(r);
}
