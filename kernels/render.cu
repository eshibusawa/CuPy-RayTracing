#include "hip/hip_runtime.h"
// This file is part of CuPy-RayTracing.
// Copyright (c) 2025, Eijiro Shibusawa <phd_kimberlite@yahoo.co.jp>
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this
//    list of conditions and the following disclaimer.
// 2. Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
// ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
// WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR
// ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
// (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
// LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
// ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include <hiprand/hiprand_kernel.h>

using color = vec3;
using point3 = vec3;

__constant__ vec3 g_cameraCenter;
__constant__ vec3 g_pixelDeltaU;
__constant__ vec3 g_pixelDeltaV;
__constant__ vec3 g_pixel00Loc;
__constant__ point3 g_defocusDiskU;
__constant__ point3 g_defocusDiskV;
__constant__ world *g_world;

__device__ bool hit(const type_and_index *p,  ray& r, interval ray_t, hit_record& rec)
{
  bool ret = false;
  const sphere *hittable_sphere = NULL;
  switch (p->type)
  {
  case 0:
    hittable_sphere = &(g_world->spheres[p->index]);
    ret = hit(hittable_sphere, r, ray_t, rec);
    break;

  default:
    ret = false;
    break;
  }

  return ret;
}

__device__ bool scatter(const type_and_index *p, const ray& r_in, const hit_record& rec, color& attenuation, ray& scattered, hiprandStateXORWOW_t &randomState)
{
  bool ret = false;
  const lambertian *material_lambertian = NULL;
  const metal *material_metal = NULL;
  const dielectric *material_dielectric = NULL;

  switch (p->type)
  {
  case 0:
    material_lambertian = &(g_world->lambertians[p->index]);
    ret = scatter(material_lambertian, r_in, rec, attenuation, scattered, randomState);
    break;

  case 1:
    material_metal = &(g_world->metals[p->index]);
    ret = scatter(material_metal, r_in, rec, attenuation, scattered, randomState);
    break;

  case 2:
    material_dielectric = &(g_world->dielectrics[p->index]);
    ret = scatter(material_dielectric, r_in, rec, attenuation, scattered, randomState);
    break;

  default:
    ret = false;
    break;
  }

  return ret;
}

__device__ color ray_color(const ray& r, const hittable_list& world, hiprandStateXORWOW_t &randomState)
{
  ray cur_ray = r;
  color cur_attenuation = color(1.f, 1.f, 1.f);

  for(int i = 0; i < (RTOW_MAX_DEPTH); i++)
  {
    hit_record rec;
    if (hit(&world, cur_ray, interval(0.001f, RTOW_FLT_MAX), rec))
    {
      ray scattered;
      color attenuation;
      if (scatter(&(rec.material_ti), cur_ray, rec, attenuation, scattered, randomState))
      {
        cur_attenuation = cur_attenuation * attenuation;
        cur_ray = scattered;
      }
      else
      {
        return vec3(0, 0, 0);
      }
    }
    else
    {
      vec3 unit_direction = unit_vector(cur_ray.direction());
      float a = 0.5f * (unit_direction.y() + 1.0f);
      color c = (1.0f - a) * vec3(1.0f, 1.0f, 1.0f) + a *vec3(0.5f, 0.7f, 1.0f);
      return cur_attenuation * c;
    }
  }

  return vec3(0, 0, 0);
}

__device__ vec3 sample_square(hiprandStateXORWOW_t &randomState)
{
  // Returns the vector to a random point in the [-.5,-.5]-[+.5,+.5] unit square.
  return vec3(hiprand_uniform(&randomState) - 0.5f, hiprand_uniform(&randomState) - 0.5f, 0);
}

__device__ point3 defocus_disk_sample(hiprandStateXORWOW_t &randomStat)
{
  // Returns a random point in the camera defocus disk.
  auto p = random_in_unit_disk(randomStat);
  return g_cameraCenter + (p.x() * g_defocusDiskU) + (p.y() * g_defocusDiskV);
}

__device__ ray get_ray(int i, int j, hiprandStateXORWOW_t &randomState)
{
  // Construct a camera ray originating from the origin and directed at randomly sampled
  // point around the pixel location i, j.
  auto offset = sample_square(randomState);
  auto pixelSample = g_pixel00Loc + ((offset.x() + i) * g_pixelDeltaU) + ((offset.y() + j) * g_pixelDeltaV);
  auto rayOrigin = ((RTOW_DEFOCUS_ANGLE) <= 0) ? g_cameraCenter : defocus_disk_sample(randomState);
  auto rayDirection = pixelSample - rayOrigin;
  ray r(rayOrigin, rayDirection);

  return r;
}

extern "C" __global__ void render(vec3 *output, unsigned long long randomState)
{
  const int indexX = threadIdx.x + blockIdx.x * blockDim.x;
  const int indexY = threadIdx.y + blockIdx.y * blockDim.y;
  if((indexX >= (RTOW_WIDTH)) || (indexY >= (RTOW_HEIGHT)))
  {
    return;
  }

  const int index = indexY * (RTOW_WIDTH) + indexX;
  hiprandStateXORWOW_t lrs;
  hiprand_init(randomState, index, 0, &lrs);
  color pixel_color(0, 0, 0);
  #pragma unroll
  for (int sample = 0; sample < (RTOW_SAMPLES_PER_PIXEL); sample++)
  {
    ray r = get_ray(indexX, indexY, lrs);
    pixel_color += ray_color(r, *(g_world->hittable_lists), lrs);
  }
  pixel_color *= (RTOW_PIXEL_SAMPLE_SCALE);
  const interval intensity(0.f, 0.999f);
  output[index] = color(intensity.clamp(pixel_color.x()), intensity.clamp(pixel_color.y()), intensity.clamp(pixel_color.z()));
}
